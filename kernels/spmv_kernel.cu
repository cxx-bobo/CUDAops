#include "hip/hip_runtime.h"
#include <spmv.cuh>
#include <cassert>
#include <iostream>

#define FULL_WARP_MASK 0xffffffff
#define NNZ_PER_WG 64u  ///u表示无符号整数


//csr_spmv_scalar_kernel
__global__ void csr_spmv_scalar_kernel (
    const uint64_t n_rows,
    const uint64_t *col_ids,
    const uint64_t *row_ptr,
    const float *data,
    const float *x,
    float *y)
{
    uint64_t row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n_rows)
    {
        const uint64_t row_start = row_ptr[row];
        const uint64_t row_end = row_ptr[row + 1];
        float sum = 0;
        for (uint64_t element = row_start; element< row_end; element++){
            sum += data[element] * x[col_ids[element]];
        }
        y[row] = sum;
    }
}


//csr_spmv_vector_kernel
  ///sum reduction
__device__ float warp_reduce (float val)
{
  for (int offset = warpSize / 2; offset > 0; offset /= 2)
     val += __shfl_down_sync (FULL_WARP_MASK,val,offset);
  return val;
}

__global__ void csr_spmv_vector_kernel (
  const uint64_t n_rows,
  const uint64_t *col_ids,
  const uint64_t *row_ptr,
  const float *data,
  const float *x,
  float *y)
{
  const uint64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  const uint64_t warp_id = thread_id / 32;
  const uint64_t lane = thread_id % 32;
  const uint64_t row = warp_id; ///< One warp per row
  float sum =0;
  if (row < n_rows)
 {
   const uint64_t row_start = row_ptr[row];
   const uint64_t row_end = row_ptr[row + 1];
   for (uint64_t element = row_start + lane; element < row_end; element += 32)
       sum += data[element] * x[col_ids[element]];
  }
 sum = warp_reduce (sum);
 if (lane == 0 && row < n_rows)
    y[row] = sum;
}


//csr_spmv_adaptive_kernel
template <typename data_type>
__global__ void fill_vector (unsigned int n, data_type *vec, data_type value)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n)
    vec[i] = value;
}

__device__ unsigned int prev_power_of_2 (unsigned int n)
{
  while (n & n - 1)
    n = n & n - 1;
  return n;
}

template <typename data_type>
__global__ void csr_spmv_adaptive_kernel (
    const unsigned int n_rows,
    const unsigned int *col_ids,
    const unsigned int *row_ptr,
    const unsigned int *row_blocks,
    const data_type *data,
    const data_type *x,
    data_type *y)
{
  const unsigned int block_row_begin = row_blocks[blockIdx.x];
  const unsigned int block_row_end = row_blocks[blockIdx.x + 1];
  const unsigned int nnz = row_ptr[block_row_end] - row_ptr[block_row_begin];

  __shared__ data_type cache[NNZ_PER_WG];

  if (block_row_end - block_row_begin > 1)
  {
    /// CSR-Stream case
    const unsigned int i = threadIdx.x;
    const unsigned int block_data_begin = row_ptr[block_row_begin];
    const unsigned int thread_data_begin = block_data_begin + i;

    if (i < nnz)
      cache[i] = data[thread_data_begin] * x[col_ids[thread_data_begin]];
    __syncthreads ();

    const unsigned int threads_for_reduction = prev_power_of_2 (blockDim.x / (block_row_end - block_row_begin));

    if (threads_for_reduction > 1)
      {
        /// Reduce all non zeroes of row by multiple thread
        const unsigned int thread_in_block = i % threads_for_reduction;
        const unsigned int local_row = block_row_begin + i / threads_for_reduction;

        data_type dot = 0.0;

        if (local_row < block_row_end)
          {
            const unsigned int local_first_element = row_ptr[local_row] - row_ptr[block_row_begin];
            const unsigned int local_last_element = row_ptr[local_row + 1] - row_ptr[block_row_begin];

            for (unsigned int local_element = local_first_element + thread_in_block;
                 local_element < local_last_element;
                 local_element += threads_for_reduction)
              {
                dot += cache[local_element];
              }
          }
        __syncthreads ();
        cache[i] = dot;

        /// Now each row has threads_for_reduction values in cache
        for (int j = threads_for_reduction / 2; j > 0; j /= 2)
          {
            /// Reduce for each row
            __syncthreads ();

            const bool use_result = thread_in_block < j && i + j < NNZ_PER_WG;

            if (use_result)
              dot += cache[i + j];
            __syncthreads ();

            if (use_result)
              cache[i] = dot;
          }

        if (thread_in_block == 0 && local_row < block_row_end)
          y[local_row] = dot;
      }
    else
      {
        /// Reduce all non zeroes of row by single thread
        unsigned int local_row = block_row_begin + i;
        while (local_row < block_row_end)
          {
            data_type dot = 0.0;

            for (unsigned int j = row_ptr[local_row] - block_data_begin;
                 j < row_ptr[local_row + 1] - block_data_begin;
                 j++)
              {
                dot += cache[j];
              }

            y[local_row] = dot;
            local_row += NNZ_PER_WG;
          }
      }
  }
  else
  {
    const unsigned int row = block_row_begin;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane = threadIdx.x % 32;

    data_type dot = 0;

    if (nnz <= 64 || NNZ_PER_WG <= 32)
    {
      /// CSR-Vector case
      if (row < n_rows)
      {
        const unsigned int row_start = row_ptr[row];
        const unsigned int row_end = row_ptr[row + 1];

        for (unsigned int element = row_start + lane; element < row_end; element += 32)
          dot += data[element] * x[col_ids[element]];
      }

      dot = warp_reduce (dot);

      if (lane == 0 && warp_id == 0 && row < n_rows)
      {
        y[row] = dot;
      }
    }
    else
    {
      /// CSR-VectorL case
      if (row < n_rows)
      {
        const unsigned int row_start = row_ptr[row];
        const unsigned int row_end = row_ptr[row + 1];

        for (unsigned int element = row_start + threadIdx.x; element < row_end; element += blockDim.x)
          dot += data[element] * x[col_ids[element]];
      }

      dot = warp_reduce (dot);

      if (lane == 0)
        cache[warp_id] = dot;
      __syncthreads ();

      if (warp_id == 0)
      {
        dot = 0.0;

        for (unsigned int element = lane; element < blockDim.x / 32; element += 32)
          dot += cache[element];

        dot = warp_reduce (dot);

        if (lane == 0 && row < n_rows)
        {
          y[row] = dot;
        }
      }
    }
  }
}

unsigned int fill_row_blocks (
  bool fill,
  unsigned int rows_count,
  const unsigned int *row_ptr,
  unsigned int *row_blocks
)
{
  if (fill)
    row_blocks[0] = 0;

  int last_i = 0;
  int current_wg = 1;
  unsigned int nnz_sum = 0;
  for (int i = 1; i <= rows_count; i++)
  {
    nnz_sum += row_ptr[i] - row_ptr[i - 1];

    if (nnz_sum == NNZ_PER_WG)
    {
      last_i = i;

      if (fill)
        row_blocks[current_wg] = i;
      current_wg++;
      nnz_sum = 0;
    }
    else if (nnz_sum > NNZ_PER_WG)
    {
      if (i - last_i > 1)
      {
        if (fill)
          row_blocks[current_wg] = i - 1;
        current_wg++;
        i--;
      }
      else
      {
        if (fill)
          row_blocks[current_wg] = i;
        current_wg++;
      }

      last_i = i;
      nnz_sum = 0;
    }
    else if (i - last_i > NNZ_PER_WG)
    {
      last_i = i;
      if (fill)
        row_blocks[current_wg] = i;
      current_wg++;
      nnz_sum = 0;
    }
  }

  if (fill)
    row_blocks[current_wg] = rows_count;

  return current_wg;
}