#include "hip/hip_runtime.h"
#include <spmv.cuh>
#include <cassert>
#include <iostream>

#define FULL_WARP_MASK 0xffffffff

__global__ void csr_spmv_scalar_kernel (
    const uint64_t n_rows,
    const uint64_t *col_ids,
    const uint64_t *row_ptr,
    const float *data,
    const float *x,
    float *y)
{
    uint64_t row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n_rows)
    {
        const uint64_t row_start = row_ptr[row];
        const uint64_t row_end = row_ptr[row + 1];
        float sum = 0;
        for (uint64_t element = row_start; element< row_end; element++){
            sum += data[element] * x[col_ids[element]];
        }
        y[row] = sum;
        // printf("y[%d] = %f \n",row,y[row]);
    }
}


__device__ float warp_reduce (float val)
{
  for (int offset = warpSize / 2; offset > 0; offset /= 2)
     val += __shfl_down_sync (FULL_WARP_MASK,val,offset);
  return val;
}

__global__ void csr_spmv_vector_kernel (
  const uint64_t n_rows,
  const uint64_t *col_ids,
  const uint64_t *row_ptr,
  const float *data,
  const float *x,
  float *y)
{
  const uint64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  const uint64_t warp_id = thread_id / 32;
  const uint64_t lane = thread_id % 32;
  const uint64_t row = warp_id; ///< One warp per row
  float sum =0;
  if (row < n_rows)
 {
   const uint64_t row_start = row_ptr[row];
   const uint64_t row_end = row_ptr[row + 1];
   for (uint64_t element = row_start + lane; element < row_end; element += 32)
       sum += data[element] * x[col_ids[element]];
  }
 sum = warp_reduce (sum);
 if (lane == 0 && row < n_rows)
    y[row] = sum;
}