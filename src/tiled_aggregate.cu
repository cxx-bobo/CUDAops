#include "hip/hip_runtime.h"
// This program computes matrix multiplication using shared memory tiling
// By: Nick from CoffeeBeforeArch

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

#include <nvToolsExt.h>


__global__ void tiledMatrixMul(
    const int *matrix_W, 
    const int *matrix_H, 
    const int *vector_b,
    const int tile_size,
    int *c,
    const int N) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Statically allocated shared memory
  extern __shared__ int tile[];
  int* tile_W = tile;
  int* tile_H = tile+tile_size*tile_size;

  // Accumulate in temporary variable
  int tmp = 0;

  // Sweep tile across matrix
  for (int i = 0; i < N; i += blockDim.x) {
    // Load in elements for this tile
    tile_W[threadIdx.y * blockDim.x + threadIdx.x] = matrix_W[row * N + i + threadIdx.x];
    tile_H[threadIdx.y * blockDim.x + threadIdx.x] = matrix_H[i * N + threadIdx.y * N + col];

    // Wait for both tiles to be loaded in before doing computation
    __syncthreads();

    // Do matrix multiplication on the small matrix
    for (int j = 0; j < blockDim.x; j++) {
      tmp +=
          tile_W[threadIdx.y * blockDim.x + j] * tile_H[j * blockDim.x + threadIdx.x];
    }
    
    // Wait for all threads to finish using current tiles before loading in new
    // ones
    __syncthreads();
  }

  // Write back results
  tmp += vector_b[row];
  c[row * N + col] = tmp;
}

// Check result on the CPU
void verify_result(
  std::vector<int> &matrix_W, 
  std::vector<int> &matrix_H, 
  std::vector<int> &vector_b,
  std::vector<int> &c,
  int N) {
  // For every row...
  for (int i = 0; i < N; i++) {
    // For every column...
    for (int j = 0; j < N; j++) {
      // For every element in the row-column pair
      int tmp = 0;
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        tmp += matrix_W[i * N + k] * matrix_H[k * N + j];
      }
      tmp += vector_b[i];
      // Check against the CPU result
      assert(tmp == c[i * N + j]);
    }
  }
}

int main() {

  // Matrix size of 1024 x 1024;
  constexpr int N = 1 << 10;

  // Size (in bytes) of matrix
  size_t matrix_size = N * N * sizeof(int);
  size_t vector_size = N * sizeof(int);

  // Host vectors
  nvtxRangePush("allocate host memory for three matrices and one vector");
  std::vector<int> h_matrix_W(N * N);
  std::vector<int> h_matrix_H(N * N);
  std::vector<int> h_matrix_HK(N * N);
  std::vector<int> h_vector_b(N);
  nvtxRangePop();

  // Initialize matrices
  nvtxRangePush("initialize two source matrices and one vectorwith random numbers");
  std::generate(h_matrix_W.begin(), h_matrix_W.end(), []() { return rand() % 100; });
  std::generate(h_matrix_H.begin(), h_matrix_H.end(), []() { return rand() % 100; });
  std::generate(h_vector_b.begin(), h_vector_b.end(), []() { return rand() % 100; });
  // std::cout << "vector_b = " << h_vector_b;
  nvtxRangePop();

  // Allocate device memory
  nvtxRangePush("allocate device memory for three matrices and one vector");
  int *d_matrix_W, *d_matrix_H, *d_vector_b, *d_matrix_HK;
  hipMalloc(&d_matrix_W, matrix_size);
  hipMalloc(&d_matrix_H, matrix_size);
  hipMalloc(&d_matrix_HK, matrix_size);
  hipMalloc(&d_vector_b, vector_size);
  nvtxRangePop();

  // Copy data to the device
  nvtxRangePush("copy matrices and vector from host to device memory");
  hipMemcpy(d_matrix_W, h_matrix_W.data(), matrix_size, hipMemcpyHostToDevice);
  hipMemcpy(d_matrix_H, h_matrix_H.data(), matrix_size, hipMemcpyHostToDevice);
  hipMemcpy(d_vector_b, h_vector_b.data(), vector_size, hipMemcpyHostToDevice);
  nvtxRangePop();

  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = N / THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // obtain shared memory size for each thread block(tile_A+tile_B,所以乘2)
  int shared_memory_size = 2*THREADS*THREADS*sizeof(int);

  // Launch kernel
  std::cout << "Launch Kernel: " << THREADS << " threads per block, " << BLOCKS << " blocks in the grid" << std::endl;
  nvtxRangePush("start kernel");
  tiledMatrixMul<<<blocks, threads, shared_memory_size>>>(d_matrix_W, d_matrix_H, d_vector_b, THREADS, d_matrix_HK, N);
  nvtxRangePop();

  // Copy back to the host
  nvtxRangePush("copy matrix from device to host memory");
  hipMemcpy(h_matrix_HK.data(), d_matrix_HK, matrix_size, hipMemcpyDeviceToHost);
  nvtxRangePop();

  // Check result
  nvtxRangePush("verify result");
  verify_result(h_matrix_W, h_matrix_H, h_vector_b, h_matrix_HK, N);
  nvtxRangePop();

  std::cout << "COMPLETED SUCCESSFULLY\n";

  // Free memory on device
  nvtxRangePush("free device memory");
  hipFree(d_matrix_W);
  hipFree(d_matrix_H);
  hipFree(d_matrix_HK);
  hipFree(d_vector_b);
  nvtxRangePop();

  return 0;
}
