#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <spmv.cuh>

#include <nvToolsExt.h>

void generateSparseMatrix(
    const uint64_t numRows, 
    const uint64_t numCols, 
    double density,
    std::vector<float> &values, 
    std::vector<uint64_t> &colIndices, 
    std::vector<uint64_t> &rowOffsets
);

void verifySpMVresult(
    const uint64_t numRows, 
    const uint64_t numCols, 
    std::vector<float> &values, 
    std::vector<uint64_t> &col_idx, 
    std::vector<uint64_t> &row_ptr,
    std::vector<float> &x,
    std::vector<float> &y
);

int main() {
  // initial constants
  constexpr uint64_t numRows = 1<<4; 
  constexpr uint64_t sizeRow = 1<<3;
  const double density = 0.1;
  
  size_t size_x = sizeRow * sizeof(float);
  size_t size_y = numRows * sizeof(float);
  
  // Host vectors 
  std::vector<float> values; 
  std::vector<uint64_t> col_idx; 
  std::vector<uint64_t> row_ptr;
  std::vector<float> h_x;
  h_x.reserve(sizeRow);
  std::vector<float> h_y;
  h_y.reserve(numRows);
  
  // Initialize csr and vector_x
  nvtxRangePush("initialize csr and vector with random numbers");
  generateSparseMatrix(numRows, sizeRow, density, values, col_idx, row_ptr);
  for (int i=0; i<sizeRow; i++){
    h_x.push_back(static_cast<float>(rand() % 100));
  };
  nvtxRangePop();

  // Allocate device memory
  nvtxRangePush("allocate device memory");
  float *d_values, *d_x, *d_y;
  uint64_t *d_col_idx, *d_row_ptr;
  hipMalloc(&d_values, sizeof(float)*values.size());
  hipMalloc(&d_col_idx, sizeof(uint64_t)*col_idx.size());
  hipMalloc(&d_row_ptr, sizeof(uint64_t)*row_ptr.size());
  hipMalloc(&d_x, size_x);
  hipMalloc(&d_y, size_y);
  nvtxRangePop();

  // Copy data to the device
  nvtxRangePush("copy data from host to device memory");
  hipMemcpy(d_values, values.data(), sizeof(float)*values.size(), hipMemcpyHostToDevice);
  hipMemcpy(d_col_idx, col_idx.data(), sizeof(uint64_t)*col_idx.size(), hipMemcpyHostToDevice);
  hipMemcpy(d_row_ptr, row_ptr.data(), sizeof(uint64_t)*row_ptr.size(), hipMemcpyHostToDevice);
  hipMemcpy(d_x, h_x.data(), size_x, hipMemcpyHostToDevice);
  nvtxRangePop();

  // Threads per CTA dimension
  int threads_per_CTAdim = 1<<7;

  // Blocks per grid dimension (1行1个warp，1个warp有32个threads)
  int blocks_per_GRIDdim = numRows*32 / threads_per_CTAdim;

  // Launch kernel
  std::cout << "Launch Kernel: " << threads_per_CTAdim << " threads per block, " << blocks_per_GRIDdim << " blocks in the grid" << std::endl;
  nvtxRangePush("Launch kernel");
  csr_spmv_vector_kernel<<<blocks_per_GRIDdim, threads_per_CTAdim>>>(numRows, d_col_idx, d_row_ptr, d_values, d_x, d_y);
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess){
    printf("kernel launch failed with error \"%s\".\n",
    hipGetErrorString(cudaerr));
    exit(-1);
  }  
  nvtxRangePop();

  // Copy back to the host
  nvtxRangePush("copy data from device to host memory");
  hipMemcpy(h_y.data(), d_y, size_y, hipMemcpyDeviceToHost);
  nvtxRangePop();

  // Check result
  nvtxRangePush("veryfy result");
  verifySpMVresult(numRows, sizeRow, values, col_idx, row_ptr, h_x, h_y);
  nvtxRangePop();

  // Free memory on device
  nvtxRangePush("free device memory");
  hipFree(d_values);
  hipFree(d_col_idx);
  hipFree(d_row_ptr);
  hipFree(d_x);
  hipFree(d_y);
  nvtxRangePop();

  std::cout << "\ncsr_spmv_scalar COMPLETED SUCCESSFULLY\n";

  return 0;
}
